/**
 * Copyright (c) 2020, Guillermo G. Trabes
 * Carleton University, Universidad Nacional de San Luis
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef GPU_PARALLEL_CUDA_CU
#define GPU_PARALLEL_CUDA_CU

#include "gpu_parallel_cuda.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <iostream>
#include <sched.h>
#include <vector>
#include <utility>
#include <tuple>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//namespace gpu_parallel_openmp {

	template<class T, class Function>
	void gpu_parallel_for_each_object(std::vector<T>& obj, Function& f, unsigned int thread_number){
		/* set number of threads */
		//omp_set_num_threads(thread_number);
		size_t size = obj.size();

		#pragma omp parallel for num_threads(thread_number) firstprivate(f) shared(obj)
		for(size_t i = 0; i < size; i++){
			f(obj[i]);
		}

	}

	template<typename ITERATOR, typename FUNC>
	void gpu_parallel_for_each_iterator(ITERATOR first, ITERATOR last, FUNC& f, unsigned int thread_number){
		/* set number of threads */
		//omp_set_num_threads(thread_number);
		size_t n = std::distance(first, last);

		//#pragma omp parallel for num_threads(thread_number) firstprivate(f, first)
		//for(int i = 0; i < n; i++){
		//	f(*(i+first));
		//}

		#pragma omp target
                for(int i = 0; i < n; i++){
                        f(*(i+first));
                }

//		#pragma omp parallel for firstprivate(f) shared(first,last)
//    	for (ITERATOR it = first; it != last; it++) {
//    		f(*it);
//    	}

//		#pragma omp parallel for firstprivate(f) shared(first)
//		for(size_t i = 0; i < n; i++){
//			auto& elem = *(first + i);
			// do whatever you want with elem
//			f(elem);
//		}
	}
	
	
	void printCudaVersion()
	{
    	std::cout << "CUDA Compiled version: " << __CUDACC_VER_BUILD__ << std::endl;

    	int runtime_ver;
    	hipRuntimeGetVersion(&runtime_ver);
    	std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    	int driver_ver;
    	hipDriverGetVersion(&driver_ver);
    	std::cout << "CUDA Driver version: " << driver_ver << std::endl;
    	
    	int num_gpus = 0 ;
    	hipGetDeviceCount(&num_gpus);
    	std::cout << "CUDA Devices: " << num_gpus << std::endl;
	}
	
	

//}

#endif //GPU_PARALLEL_CUDA_CU
